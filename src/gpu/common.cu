#include "hip/hip_runtime.h"

namespace gccl {

void GCCLSetCudaDevice(int dev_id) { hipSetDevice(dev_id); }

}  // namespace gccl